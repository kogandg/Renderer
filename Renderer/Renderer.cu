#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "CImg.h"
using namespace cimg_library;

#include <iostream>
#include <time.h>


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__ void render(float* frameBuffer, int maxX, int maxY)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= maxX) || (y >= maxY))
	{
		return;
	}

	int pixelIndex = y * maxX * 3 + x * 3;
	frameBuffer[pixelIndex + 0] = float(x) / maxX;
	frameBuffer[pixelIndex + 1] = float(y) / maxY;
	frameBuffer[pixelIndex + 2] = 0.2;
}

void hostRender(float* frameBuffer, int maxX, int maxY)
{
	for (int y = 0; y < maxY; y++)
	{
		for (int x = 0; x < maxX; x++)
		{
			int pixelIndex = y * maxX * 3 + x * 3;
			frameBuffer[pixelIndex + 0] = float(x) / maxX;
			frameBuffer[pixelIndex + 1] = float(y) / maxY;
			frameBuffer[pixelIndex + 2] = 0.2;
		}
	}
}

void saveFrameBufferToImage(std::string fileName, float* frameBuffer, int imageX, int imageY)
{
	CImg<unsigned char> image(imageX, imageY, 1, 3, 0);
	for (int y = 0; y < imageY; y++)
	{
		for (int x = 0; x < imageX; x++)
		{
			int pixelIndex = y * 3 * imageX + x * 3;
			unsigned char color[3];
			color[0] = 256 * frameBuffer[pixelIndex];
			color[1] = 256 * frameBuffer[pixelIndex + 1];
			color[2] = 256 * frameBuffer[pixelIndex + 2];
			image.draw_point(x, y, color);
		}
	}
	image.save("fileName");
}

int main()
{
	int* cudaDevices = 0;
	hipGetDeviceCount(cudaDevices);

	int imageX = 1200;
	int imageY = 600;
	int threadX = 8;
	int threadY = 8;

	int numPixels = imageX * imageY;
	size_t frameBufferSize = 3 * numPixels * sizeof(float);

	float* frameBuffer;
	
	clock_t start;
	clock_t stop;

	if (cudaDevices == 0)
	{
		std::cout << "No cuda devices" << std::endl;

		frameBuffer = new float[frameBufferSize];

		start = clock();
		
		hostRender(frameBuffer, imageX, imageY);

		stop = clock();
	}
	else
	{
		checkCudaErrors(hipMallocManaged((void**)&frameBuffer, frameBufferSize));

		start = clock();

		dim3 blocks(imageX / threadX + 1, imageY / threadY + 1);
		dim3 threads(threadX, threadY);

		render << <blocks, threads >> > (frameBuffer, imageX, imageY);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		stop = clock();
	}
	
	double timerSeconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cout << "Rendering took " << timerSeconds << " seconds" << std::endl;

	saveFrameBufferToImage("out.bmp", frameBuffer, imageX, imageY);

	if (cudaDevices == 0)
	{
		delete frameBuffer;
	}
	else
	{
		checkCudaErrors(hipFree(frameBuffer));
	}
}